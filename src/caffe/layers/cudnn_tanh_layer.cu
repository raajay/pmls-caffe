
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNTanHLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
      CUDNN_ACTIVATION_TANH,
      cudnn::dataType<Dtype>::one,
      this->bottom_desc_, bottom_data,
      cudnn::dataType<Dtype>::zero,
      this->top_desc_, top_data));
}

template <typename Dtype>
void CuDNNTanHLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = (*bottom)[0]->gpu_data();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
      CUDNN_ACTIVATION_TANH,
      cudnn::dataType<Dtype>::one,
      this->top_desc_, top_data, this->top_desc_, top_diff,
      this->bottom_desc_, bottom_data,
      cudnn::dataType<Dtype>::zero,
      this->bottom_desc_, bottom_diff));
}

INSTANTIATE_CLASS(CuDNNTanHLayer);

}  // namespace caffe
#endif